#include "hip/hip_runtime.h"
#include "seerac-ellis.hpp"

#include <cstdio>
#include <string>

// Thread block size
#define BLOCKSIZE 1024

#define delta 0.01
#define MCZ 1
#define HAB 2
#define WIND 3
#define COAST 4

#define DATA_PERIOD 1
#define HOLDOUT_PERIOD 2
#define FORECAST_PERIOD 3

//Lookups table ordering
#define PERIODTAB 0
#define REGTAB 1
#define COMPETETAB 2
#define SEASONSPECIESGEARDEPLETION 3 //Ordered to be season on x and species, gear on y
//Held as int, needs to be divided by 65536 to get float.

//Cell Float Data table ordering
#define STARTCOND 0
#define DATAPERIODGEARDEP 1
#define SEASONGEARDEP 2
#define SEASONSPECIESRECOVERY 3
#define HEATMAP 4
#define NUMFLOATTABS 5


__constant__ int d_lookup[MAXLOOKUPWIDTH * MAXLOOKUPHEIGHT];
__constant__ int d_offset[MAXOFFSETS];


__device__ double getSharedFloat(DataMatrix cellData, int cellIdx, int colId)
{
  unsigned long cellRef = cellIdx * cellData.width + colId;
  return cellData.device[cellRef];
}

__device__ void setSharedFloat(ResultsMatrix results, int cellIdx, int periodIdx, int speciesIdx,
  int aryLen, const int *const offset, double* value)
{
  unsigned long cellRef = (cellIdx * results.width) + (periodIdx * offset[NUMSPECIES] * NUMVALUES) +
    (speciesIdx * NUMVALUES);

  for (int idx = 0; idx < aryLen; idx++) {
    results.device[cellRef + idx] = value[idx];
  }
}

__device__ void getLookupData(int tabId, int idx, const int *const offset, int *row)
{
  int rowOffset = 0;

  if (tabId == REGTAB) {
    rowOffset = offset[NUMPERIODS];
  } else if (tabId == COMPETETAB) {
    rowOffset = offset[NUMPERIODS] + offset[NUMREGS];
  } else if (tabId == SEASONSPECIESGEARDEPLETION) {
    rowOffset = offset[NUMPERIODS] + offset[NUMREGS] + offset[NUMCOMPETES];
  }

  unsigned long cellRef = (rowOffset + idx) * offset[LOOKUPWIDTH];
  for (int idx = 0; idx < offset[LOOKUPWIDTH]; idx++) {
    row[idx] = d_lookup[cellRef + idx];
  }
}

__device__ int checkLimit(double dist, double limit)
{
  return (dist <= limit) ? 1 : 0;
}

__device__ int checkBoolean(int val, int mask)
{
  return (val && mask) ? 1 : 0;
}

__device__ int getDataColId(int tab, int colId, const int *const offset)
{
  int seps[NUMFLOATTABS] = {
    2 * offset[NUMSPECIES],
    offset[NUMGEARS] * offset[NUMDATAPERIODS],
    4 * offset[NUMGEARS],
    4 * offset[NUMSPECIES]
  };
  int res = colId;

  for (int idx = 0; idx < tab; idx++) {
    res = res + seps[idx];
  }

  return res;
}

__device__ void calcEffort(double *effortAry, int periodIdx, int cellIdx, const int *const offset,
  DataMatrix cellData)
{
  /* Returns an array of effort for each gear for a given cell at a given period when all
   * applicable restrictions and displacements applied.
   */
  int thisReg[MAXLOOKUPWIDTH];
  int thisPeriod[MAXLOOKUPWIDTH];
  double multiplier = 1.0F;

  getLookupData(PERIODTAB, periodIdx, offset, thisPeriod);

  int periodSeason = thisPeriod[1];

  int bitMask = getSharedFloat(cellData, cellIdx, getDataColId(HEATMAP,0, offset));
  double cellDist = getSharedFloat(cellData, cellIdx, getDataColId(HEATMAP,1, offset));
  int areaCode = getSharedFloat(cellData, cellIdx, getDataColId(HEATMAP,2, offset));
  int cellHab = areaCode & (0x000F);

  for (int idx = 0; idx < offset[NUMGEARS]; idx++) {
    effortAry[idx]=1.0F;
  }

  int areaType, isCoast, isMCZ, isWind, isHab;
  for (int regIdx=0; regIdx < offset[NUMREGS]; regIdx++) {
    multiplier = 1.0F;
    getLookupData(REGTAB, regIdx, offset, thisReg);

    if ((bitMask && (2 ^ (thisReg[1] - 1)) > 0)
      && ((periodSeason == 0) || (periodSeason == thisReg[10]))
      && (periodIdx >= thisReg[2]) && (periodIdx <= thisReg[3])) {

      areaType = thisReg[4];
      isCoast = (areaType == COAST) ? checkLimit(cellDist, thisReg[7]) : 0;
      isMCZ   = (areaType == MCZ)   ? checkBoolean(areaCode, 0x0010)   : 0;
      isWind  = (areaType == WIND)  ? checkBoolean(areaCode, 0x0020)   : 0;
      isHab   = ((areaType == HAB) && (cellHab == thisReg[9])) ? 2 : 0;

      if ((thisReg[2] == 2) && ((isCoast > 0) || (isHab > 0))) {
        multiplier = thisReg[6] / 256.0F; // Used to keep whole table as integers
      } else if ((isCoast > 0) || (isMCZ > 0) || (isWind > 0) || (isHab > 0)) {
        multiplier = 0.0F;
      }

      if ((multiplier != 1.0F) && (thisReg[9] == 0)) {
        for (int idx = 0; idx < offset[NUMGEARS]; idx++) {
          effortAry[idx] = effortAry[idx] * multiplier;
        }
      } else {
        effortAry[thisReg[9]] = effortAry[thisReg[9]] * multiplier;
      }
    }
  }
}

__global__ void testEllis(DataMatrix cellData, ResultsMatrix results)
{
  int cellIdx = blockIdx.x * blockDim.x + threadIdx.x;

  int offset[MAXOFFSETS];
  for (int idx = 0; idx < MAXOFFSETS; idx++) {
    offset[idx] = d_offset[idx];
  }

  int periodRow[MAXLOOKUPWIDTH];     //Used for period data
  int regRow[MAXLOOKUPWIDTH];        //Used for regs data
  int competesRow[MAXLOOKUPWIDTH];   //Used for regs data
  int depletesRow[MAXLOOKUPWIDTH];   //Used for depletion data

  int periodIdx = cellIdx % offset[NUMPERIODS];
  int regIdx = 0;
  if (offset[NUMREGS] > 0) {
    regIdx = cellIdx % offset[NUMREGS];
  }
  int competeIdx = 0;
  if (offset[NUMCOMPETES]) {
    competeIdx = cellIdx % offset[NUMCOMPETES];
  }
  int depleteIdx = cellIdx % (offset[NUMSPECIES] * offset[NUMGEARS]);

  double dummy[] = { 1.01f, 1.05f, 3.05f, 4.5f, 5.8f, 2.3f };
  setSharedFloat(results, cellIdx, 1, 0, NUMVALUES, offset, dummy);

  //Series of tests to prove get, set and lookups are working as expected
  getLookupData(PERIODTAB, periodIdx, offset, periodRow);
  getLookupData(REGTAB, regIdx, offset, regRow);
  getLookupData(COMPETETAB, competeIdx, offset, competesRow);
  getLookupData(SEASONSPECIESGEARDEPLETION, depleteIdx, offset, depletesRow);

  double result[] = {
    (double)periodRow[0],
    (double)regRow[0],
    (double)competesRow[0],
    (double)depletesRow[0],
    cellIdx * (-1.0f),
    100
  };
  setSharedFloat(results, cellIdx, 0, 0, NUMVALUES, offset, result);

  double val0 = getSharedFloat(cellData, cellIdx, 0);
  double val1 = getSharedFloat(cellData, cellIdx, getDataColId(DATAPERIODGEARDEP,0, offset));
  double val2 = getSharedFloat(cellData, cellIdx, getDataColId(SEASONGEARDEP,0, offset));
  double val3 = getSharedFloat(cellData, cellIdx, getDataColId(SEASONSPECIESRECOVERY,0, offset));
  double val4 = getSharedFloat(cellData, cellIdx, getDataColId(HEATMAP, 0, offset));
  double val5 = getSharedFloat(cellData, cellIdx, getDataColId(HEATMAP, 2, offset));

  double result2[] = { val0, val1, val2, val3, val4, val5 };
  setSharedFloat(results, cellIdx, 2, 0, 6, offset, result2);

  double result3[] = { 123.45f, 678.901f };
  setSharedFloat(results, cellIdx, 3, offset[NUMSPECIES] - 1, 2, offset, result3);

  double result4[] = { val5, val4, val3, val2, val1, val0 };
  setSharedFloat(
    results, cellIdx, (offset[NUMPERIODS] - 1), (offset[NUMSPECIES] - 1), NUMVALUES, offset, result4
  );
}

__global__ void calcEllis(DataMatrix cellData, ResultsMatrix results)
{
  int cellIdx = blockIdx.x * blockDim.x + threadIdx.x;

  int offset[MAXOFFSETS];
  for (int idx = 0; idx < MAXOFFSETS; idx++) {
    offset[idx] = d_offset[idx];
  }

  double r = 0.0F,
        dbdt = 0.0F,
        dVal = 1.0F,
        depletion = 0.0F,
        pings = 0.0F,
        sc_n = 0.0F,
        sc_n1 = 0.0F;
  double effortAry[MAXLOOKUPWIDTH];
  int periodRow[MAXLOOKUPWIDTH];   //Used for period data
  int seasonDepRow[MAXLOOKUPWIDTH];  //Used for season depletion data

  for (int periodIdx = 0; periodIdx < offset[NUMPERIODS]; periodIdx++) {
    getLookupData(PERIODTAB, periodIdx, offset, periodRow);
    int season = periodRow[2];

    for (int speciesIdx = 0; speciesIdx < offset[NUMSPECIES]; speciesIdx++) {
      sc_n = (periodIdx == 0) ? getSharedFloat(cellData, cellIdx, speciesIdx) : sc_n1;
      dVal = 1.0F;

      for (int gearIdx = 0; gearIdx < offset[NUMGEARS]; gearIdx++) {
        calcEffort(effortAry, periodIdx, cellIdx, offset, cellData);
        getLookupData(
          SEASONSPECIESGEARDEPLETION, speciesIdx * offset[NUMGEARS] + gearIdx, offset, seasonDepRow
        );
        if (periodRow[1] == 1) {
          pings = getSharedFloat(
            cellData,
            cellIdx,
            getDataColId(DATAPERIODGEARDEP, periodIdx * offset[NUMGEARS] + gearIdx, offset)
          );
        } else {
          pings = getSharedFloat(
            cellData,
            cellIdx,
            getDataColId(SEASONGEARDEP, offset[NUMGEARS] * (season - 1) + gearIdx, offset)
          );
          pings *= effortAry[gearIdx];
        }
        depletion = seasonDepRow[season] / 65536;
        dVal = dVal * pow(depletion, pings);
      } // for gearIdx

      double recRate = getSharedFloat(
        cellData,
        cellIdx,
        getDataColId(SEASONSPECIESRECOVERY, offset[NUMSPECIES] * (season - 1) + speciesIdx, offset)
      );
      double capacity = getSharedFloat(cellData, cellIdx, offset[NUMSPECIES] + speciesIdx);

      //Apply the logistic equation to the starting capacity * depletion and add recovery.....
      double dep = sc_n * dVal;
      dep = dep > 1 ? 1 : dep;

      if (dep > delta)  {
        double x0 = -logf(capacity / dep - 1) / recRate;
        r = capacity / (1 + expf(-recRate * (x0 + 1))) - dep;
        dbdt = sc_n * dVal + r - sc_n;
        sc_n1 = sc_n + dbdt;
      } else if (dep < delta) {
        dVal = 0;
        r = 0;
        dbdt = 0;
        sc_n1 = delta;
      }

      double result[] = { sc_n, dVal, r, dbdt, 0, sc_n1 };
      setSharedFloat(results, cellIdx, periodIdx, speciesIdx, NUMVALUES, offset, result);
    } // for speciesIdx

    for (int competeIdx = 0; competeIdx < offset[NUMCOMPETES]; competeIdx++) {
      /*TO DO - run thru all of the species and assess if they are competing inter species.
      Amortise appropriately use the logistic equation.....
      Probably move the dbdt calc to here as well */
    }
  } // for PeriodIdx
  __syncthreads();
}





#define CU_ERROR_THROWN(cu_error) cuErrorThrown((cu_error), __FILE__, __LINE__)
int cuErrorThrown(hipError_t cu_errno, const char *const file, int const line)
{
  if (cu_errno != hipSuccess) {
    fprintf(
      stderr,
      "CUDA ERROR: %s:%d code=%d(%s) \"%s\"\n",
      file, line, (unsigned int)cu_errno, hipGetErrorName(cu_errno), hipGetErrorString(cu_errno)
    );
    hipDeviceReset();
    return 1;
  }
  return 0;
}

int logCudaMemory() {
  double free_m, total_m, used_m;
  size_t free_t, total_t;

  if (CU_ERROR_THROWN(hipMemGetInfo(&free_t, &total_t))) {
    return 1;
  }

  free_m = free_t / (1024.0 * 1024.0);
  total_m = total_t / (1024.0 * 1024.0);
  used_m = total_m - free_m;

  printf("MEM: mem free %.2f MB; mem total %.2f MB; mem used %.2f MB\n", free_m, total_m, used_m);

  return 0;
}

int hostCalcEllis(int mode, const int *const offset, const int *const lookup, DataMatrix cellData,
  ResultsMatrix results, int debug)
{
  if (debug > 1) {
    printf("Test or real mode: %i\n", mode);
  }

  if (debug) {
    if (logCudaMemory()) {
      return -2;
    }
  }

  if (CU_ERROR_THROWN(hipMemcpyToSymbol(HIP_SYMBOL(d_offset), offset, MAXOFFSETS * sizeof(int)))) {
    return -2;
  }

  size_t lookupSize = offset[LOOKUPWIDTH] * offset[LOOKUPHEIGHT] * sizeof(int);
  if (CU_ERROR_THROWN(hipMemcpyToSymbol(HIP_SYMBOL(d_lookup), lookup, lookupSize))) {
    return -2;
  }

  size_t inDataSize = cellData.width * cellData.height * sizeof(double);
  if (CU_ERROR_THROWN(hipMalloc(&cellData.device, inDataSize))) {
    return -2;
  }
  if (CU_ERROR_THROWN(hipMemcpy(
    cellData.device, cellData.host, inDataSize, hipMemcpyHostToDevice
    ))) {
    return -2;
  }

  // Allocate results in device memory
  if (debug > 1) {
    printf(
      "Results sized by: numPeriods: %i, numSpecies: %i, numValues: %i\n",
      offset[NUMPERIODS], offset[NUMSPECIES], NUMVALUES
    );
    printf("Mem allocated for results. ncols: %i, nrows: %i\n", results.width, results.height);
  }
  size_t resultsSize = results.width * results.height * sizeof(double);
  if (CU_ERROR_THROWN(hipMalloc(&results.device, resultsSize))) {
    return -2;
  }


  if (debug) {
    if (logCudaMemory()) {
      return -2;
    }
  }


  if (debug) {
    printf("Running GPU call\n");
  }

  int numBlocks = ceil(cellData.height / BLOCKSIZE);
  if (mode == TEST_MODE) {
    testEllis<<<numBlocks, BLOCKSIZE>>>(cellData, results);
  } else {
    calcEllis<<<numBlocks, BLOCKSIZE>>>(cellData, results);
  }

  if (debug) {
    printf("GPU complete\n");
  }


  if (CU_ERROR_THROWN(hipMemcpy(
      results.host, results.device, resultsSize, hipMemcpyDeviceToHost
    ))) {
    return -2;
  }

  hipFree(cellData.device);
  hipFree(results.device);

  if (debug > 1) {
    printf("Processing complete\n");
  }

  return 0;
}
